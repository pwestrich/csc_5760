#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#include "getRealTime.h"

#define BLOCK_SIZE 16

struct Matrix {

	float *data;
	int64_t size;
	int64_t numElements; 

};

__global__ void gpuMatrixMultiply(float *L, float *R, float *S, const int64_t size){

	//allocate some space
	float value = 0.0;

	//calculate my row and column
	const int64_t row = blockIdx.y * blockDim.y + threadIdx.y; 
	const int64_t col = blockIdx.x * blockDim.x + threadIdx.x; 

	//do not go over the array
	if (row >= size || col >= size) return;

	//sum up the terms
	for (int64_t i = 0; i < size; ++i){

		value += L[row * size + i] * R[i * size + col];

	}

	//set solution
	S[row * size + col] = value;

}

void matrixMultiply(const Matrix L, const Matrix R, const Matrix S){

	//first, allocate space on the GPU for each array
	float *gpuL = NULL;
	float *gpuR = NULL;
	float *gpuS = NULL;

	hipError_t error = hipMalloc(&gpuL, L.numElements * sizeof(float));
	if (error){

		printf("Error allocating matrix L on GPU: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);

	}

	error = hipMalloc(&gpuR, R.numElements * sizeof(float));
	if (error){

		printf("Error allocating matrix R on GPU: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);

	}

	error = hipMalloc(&gpuS, S.numElements * sizeof(float));
	if (error){

		printf("Error allocating matrix S on GPU: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);

	}

	//now copy L and R
	error = hipMemcpy(gpuL, L.data, L.numElements * sizeof(float), hipMemcpyHostToDevice);
	if (error){

		printf("Error copying memory of L to GPU: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);

	}

	error = hipMemcpy(gpuR, R.data, R.numElements * sizeof(float), hipMemcpyHostToDevice);
	if (error){

		printf("Error copying memory of R to GPU: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);

	}

	//prepare to call GPU function
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((S.size + dimBlock.x - 1) / dimBlock.x, (S.size + dimBlock.y) / dimBlock.y);

	gpuMatrixMultiply<<<dimGrid, dimBlock>>>(gpuL, gpuR, gpuS, S.size);
	
	error = hipDeviceSynchronize();
	printf("Multiply finished with status: %s\n", hipGetErrorString(error));

	//copy S back
	error = hipMemcpy(S.data, gpuS, S.numElements * sizeof(float), hipMemcpyDeviceToHost);

	if (error){

		printf("Error copying memory of S from GPU: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);

	}

	hipFree(gpuL);
	hipFree(gpuR);
	hipFree(gpuS);

}

void usage(const char *name){

	printf("Usage: %s <size>\n", name);

}

int main(const int argc, const char **argv){

	if (argc < 2){

		usage(argv[0]);
		return EXIT_FAILURE;

	}

	//read the size of the matrix to make
	const int64_t size = atoll(argv[1]);
	const int64_t numElements = size * size;

	if (size < 0 || numElements < 0){

		printf("Error: Overflow in matrix size.\n");
		return EXIT_FAILURE;

	} else if (size < 1){

		printf("Error: Matrix size too small: %lli \n", size);

	}

	printf("Creating matricies...\n");

	const double startTime = getRealTime();

	Matrix L;
	L.size = size;
	L.numElements = numElements;
	L.data = (float*) malloc(numElements * sizeof(float));

	Matrix R;
	R.size = size;
	R.numElements = numElements;
	R.data = (float*) malloc(numElements * sizeof(float));

	Matrix S;
	S.size = size;
	S.numElements = numElements;
	S.data = (float*) malloc(numElements * sizeof(float));

	srand(NULL);

	for (int64_t i = 0; i < numElements; ++i){

		L.data[i] = rand() / 100.0;
		R.data[i] = rand() / 100.0;

	}

	const double initElapsed = getRealTime() - startTime;

	printf("Initilization complete. Tool %f seconds. \nMultiplying...\n", initElapsed);

	const double multiplyStart = getRealTime();

	matrixMultiply(L, R, S);

	const double multiplyElapsed = getRealTime() - multiplyStart;

	printf("Multiply complete. Took %f seconds.\n", multiplyElapsed);

	free(L.data);
	free(R.data);
	free(S.data);

	return 0;

}

